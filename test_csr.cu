#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <fstream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "load_matrix.h"

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (csrColInd) hipFree(csrColInd); \
    if (csrVal) hipFree(csrVal); \
    if (y) hipFree(y); \
    if (y0) hipFree(y0); \
    if (z) hipFree(z); \
    if (z0) hipFree(z0); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0)

#define HANDLE_ERROR( err ) \
if (err != hipSuccess) { \
    printf("%s in %s at linedd %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
    CLEANUP("cuda error occurred"); \
    exit(-1); \
}

#define HANDLE_CUSPARSE_ERROR( err, s ) \
if (err != HIPSPARSE_STATUS_SUCCESS) { \
    CLEANUP(s); \
    exit(-1); \
}

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int dim = std::stoi(argv[2]);
    printf("p = %f dim = %d\n", p, dim);

    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 2 << 16;
    int n = m;    
    int nnz = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostCsrRowPtr = 0;
    int* hostCsrColInd = 0;
    float* hostCsrVal = 0;
    int* csrRowPtr = 0;
    int* csrColInd = 0;
    float* csrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* y0 = 0;
    float* zHostPtr = 0;
    float* z = 0;
    float* z0 = 0;

    printf("generate random CSR matrix\n");

    // nnz = randomCSRMatrix(m, n, p, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);
    nnz = readAndFillCSRMatrix(m, n, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);

    printf("density of BSR matrix is %f\n", (nnz * 1.0) / (m * n));

    printf("gpu memory malloc and memcpy...\n");

    HANDLE_ERROR( hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&csrColInd, nnz * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&csrVal, nnz * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((m + 1) * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice) );

    HANDLE_CUSPARSE_ERROR( hipsparseCreate(&handle), "CUSPARSE Library initialization failed" );

    HANDLE_CUSPARSE_ERROR( hipsparseCreateMatDescr(&descr), "CSR Matrix descriptor initialization failed" );
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    printf("prepare y and z...\n");

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**)&z, m * dim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );

    printf("warm up...\n");
    HANDLE_ERROR( hipMalloc((void**)&y0, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**)&z0, m * dim * sizeof(float)) );
    HANDLE_ERROR( hipMemset((void*)y0, 0, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMemset((void*)z0, 0, m * dim * sizeof(float)) );
    int warnupRounds = 3;
    for (int i = 0; i < warnupRounds; ++i) {
        HANDLE_CUSPARSE_ERROR( hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                               m, dim, n, nnz, &fone, descr, csrVal, csrRowPtr, csrColInd, y0, dim, &fzero, z0, m),
                                               "warmup hipsparseScsrmm2 failed" );
    }

    printf("hipsparseScsrmm...\n");
    float totalTime = 0;
    int rounds = 10;
    for (int i = 0; i < rounds; ++i) {
        HANDLE_ERROR( hipMemset((void*)z, 0, m * dim * sizeof(float)) );
        
        float time;
        hipEvent_t start, stop;
        HANDLE_ERROR( hipEventCreate(&start) );
        HANDLE_ERROR( hipEventCreate(&stop) );
        HANDLE_ERROR( hipEventRecord(start, 0) );

        HANDLE_CUSPARSE_ERROR( hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                               m, dim, n, nnz, &fone, descr, csrVal, csrRowPtr, csrColInd, y, dim, &fzero, z, m),
                                               "hipsparseScsrmm2 failed" );

        HANDLE_ERROR( hipEventRecord(stop, 0) );
        HANDLE_ERROR( hipEventSynchronize(stop) );
        HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
        printf("[%d] csrmm cost time:  %3.10f ms \n", i, time);
        totalTime += time;  
    }
    printf("average csrmm cost time: %3.10f ms \n", totalTime / rounds);

    HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost) );

    HANDLE_CUSPARSE_ERROR( hipsparseDestroyMatDescr(descr), "Matrix descriptor destruction failed" );
    descr = 0;
    HANDLE_CUSPARSE_ERROR( hipsparseDestroy(handle), "CUSPARSE Library release of resources failed" );
    handle = 0;

    CLEANUP("end");

    return 0;
}    