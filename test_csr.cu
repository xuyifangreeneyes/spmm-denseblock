#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include "hipsparse.h"

static void handleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(-1);
    }
}

#define HANDLE_ERROR( err ) (handleError( err, __FILE__, __LINE__ ))

std::mt19937_64 gen(1234);

template<typename T>
T* vec2ptr(std::vector<T> v) {
    T* ptr = (T*) malloc(v.size() * sizeof(T));
    for (size_t i = 0; i < v.size(); ++i) {
        ptr[i] = v[i];
    }
    return ptr;
}

int randomCSRMatrix(int m, int n, float p, int** hostCsrRowPtr, int** hostCsrColInd, float** hostCsrVal, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> flip(0, 1), dist(minVal, maxVal);
    *hostCsrRowPtr = (int*) malloc((m + 1) * sizeof(int));
    int cnt = 0;
    (*hostCsrRowPtr)[0] = cnt;
    std::vector<int> indices;
    std::vector<float> vals;
    for (int i = 1; i <= m; ++i) {
        for (int j = 0; j < n; ++j) {
            if (flip(gen) < p) {
                indices.push_back(j);
                vals.push_back(dist(gen));
                ++cnt;
            }
        }
        (*hostCsrRowPtr)[i] = cnt;
    }
    *hostCsrColInd = vec2ptr(std::move(indices));
    *hostCsrVal = vec2ptr(std::move(vals));

    return cnt;
}

float* randomDenseMatrix(int n, int dim, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (csrColInd) hipFree(csrColInd); \
    if (csrVal) hipFree(csrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int dim = std::stoi(argv[2]);
    printf("p = %f dim = %d\n", p, dim);

    hipError_t cudaStat1, cudaStat2, cudaStat3;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 131072;
    int n = m;    
    int nnz = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostCsrRowPtr = 0;
    int* hostCsrColInd = 0;
    float* hostCsrVal = 0;
    int* csrRowPtr = 0;
    int* csrColInd = 0;
    float* csrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    printf("generate random CSR matrix\n");

    nnz = randomCSRMatrix(m, n, p, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);

    printf("gpu memory malloc and memcpy...\n");

    cudaStat1 = hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&csrColInd, nnz * sizeof(int));
    cudaStat3 = hipMalloc((void**)&csrVal, nnz * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (CSR matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((m + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (CSR matrix)");
        return 1;
    }

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    cudaStat1 = hipMalloc((void**)&y, n * dim * sizeof(float));
    cudaStat2 = hipMalloc((void**)&z, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Device malloc failed (dense matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (y)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z, 0, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed (z)");
        return 1;
    }

    printf("hipsparseScsrmm...\n");

    float time;
    hipEvent_t start, stop;

    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, dim, n, nnz,
                            &fone, descr, csrVal, csrRowPtr, csrColInd, y, n, &fzero, z, m);

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );

    printf("csrmm cost time:  %3.10f ms \n", time);  
    
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("csrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed (z)");
        return 1;
    }

    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    CLEANUP("end");

    return 0;
}    