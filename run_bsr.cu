#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <fstream>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "hipsparse.h"

static void handle_error(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at linedd %d\n", hipGetErrorString(err), file, line);
        exit(-1);
    }
}

#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

std::mt19937_64 gen(1234);

void readCSRMatrix(int m, int n, int nnz, int** hostCsrRowPtr, int** hostCsrColInd, float** hostCsrVal) {
    *hostCsrRowPtr = (int*) malloc((m + 1) * sizeof(int));
    *hostCsrColInd = (int*) malloc(nnz * sizeof(int));
    *hostCsrVal = (float*) malloc(nnz * sizeof(float));
    
    std::fstream s1("collab_ndmetis_indptr.txt");
    std::fstream s2("collab_ndmetis_indices.txt");
    int _m_1;
    s1 >> _m_1;
    printf("m = %d _m_1 = %d\n", m, _m_1);
    assert(m + 1 == _m_1);
    for (int i = 0; i <= m; ++i) {
        int x;
        s1 >> x;
        (*hostCsrRowPtr)[i] = x;
    }

    printf("kkkkk2\n");

    int _nnz;
    s2 >> _nnz;
    assert(nnz == _nnz);
    for (int i = 0; i < nnz; ++i) {
        int x;
        s2 >> x;
        (*hostCsrColInd)[i] = x;
        (*hostCsrVal)[i] = 1.0;
    }
}

float* randomDenseMatrix(int n, int dim, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (csrColInd) hipFree(csrColInd); \
    if (csrVal) hipFree(csrVal); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

int main() {
    hipError_t cudaStat1, cudaStat2, cudaStat3;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t csrDescr = 0, bsrDescr = 0;

    int m = 235868;
    int n = m;    
    int nnz = 2358104;
    int blockDim = 16;
    int mb = (m + blockDim - 1) / blockDim;
    int nb = (n + blockDim - 1) / blockDim;
    int nnzb = 0;
    int dim = 64;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostCsrRowPtr = 0;
    int* hostCsrColInd = 0;
    float* hostCsrVal = 0;
    int* csrRowPtr = 0;
    int* csrColInd = 0;
    float* csrVal = 0;

    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    yHostPtr = randomDenseMatrix(nb * blockDim, dim);
    zHostPtr = (float*) malloc(mb * blockDim * dim * sizeof(float));

    cudaStat1 = hipMalloc((void**)&y, nb * blockDim * dim * sizeof(float));
    cudaStat2 = hipMalloc((void**)&z, mb * blockDim * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Device malloc failed (dense matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(nb * blockDim * dim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (y)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z, 0, mb * blockDim * dim * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed (z)");
        return 1;
    }

    printf("read CSR matrix...\n");

    readCSRMatrix(m, n, nnz, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);

    printf("gpu memory malloc and memcpy...\n");

    cudaStat1 = hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int));
    printf("kkkkkn1\n");
    
    cudaStat2 = hipMalloc((void**)&csrColInd, nnz * sizeof(int));
    
    printf("kkkkkn2\n");

    cudaStat3 = hipMalloc((void**)&csrVal, nnz * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (CSR matrix)");
        return 1;
    }

    printf("kkkkk3\n");


    cudaStat1 = hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((m + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (CSR matrix)");
        return 1;
    }
    
    printf("kkkkk4\n");

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }
    
    status = hipsparseCreateMatDescr(&csrDescr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(csrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csrDescr, HIPSPARSE_INDEX_BASE_ZERO);

    status = hipsparseCreateMatDescr(&bsrDescr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("BSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(bsrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsrDescr, HIPSPARSE_INDEX_BASE_ZERO);

    printf("kkkkk5\n");


    cudaStat1 = hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed1 (BSR matrix)");
        printf("%s\n", hipGetErrorString(cudaStat1));
        return 1;
    }
    printf("kkkkk5.5\n");

    int base;
    int *nnzTotalDevHostPtr = &nnzb;

    status = hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, csrDescr, csrRowPtr, 
                                 csrColInd, blockDim, bsrDescr, bsrRowPtr, nnzTotalDevHostPtr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("hipsparseXcsr2bsrNnz failed");
        return 1;
    }
    printf("nnz = %d", nnz);
    if (NULL != nnzTotalDevHostPtr) {
        nnzb = *nnzTotalDevHostPtr;
    } else {
        hipMemcpy(&nnzb, bsrRowPtr + mb, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&base, bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost);
        nnzb -= base;
    }

    printf("kkkkk6\n");

    long long a = (long long) nnzb * (long long)(blockDim * blockDim) * sizeof(float);
    printf("aaa === %lld", a);
    cudaStat1 = hipMalloc((void**)&bsrColInd, nnzb * sizeof(int));
    cudaStat2 = hipMalloc((void**)&bsrVal, a);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Device malloc failed2 (BSR matrix)");
        printf("%s\n", hipGetErrorString(cudaStat1));
        printf("%s\n", hipGetErrorString(cudaStat2));
        return 1;
    }
    status = hipsparseScsr2bsr(handle, HIPSPARSE_DIRECTION_ROW, m, n, csrDescr, csrVal, csrRowPtr, csrColInd,
                              blockDim, bsrDescr, bsrVal, bsrRowPtr, bsrColInd);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("hipsparseScsr2bsr failed");
        return 1;
    }
    
    // if (csrVal) {
    //     cudaFree(csrVal);
    //     csrVal = 0;
    // }
    // if (csrRowPtr) {
    //     cudaFree(csrRowPtr);
    //     csrRowPtr = 0;
    // }
    // if (csrColInd) {
    //     cudaFree(csrColInd);
    //     csrColInd = 0;
    // }

    printf("density:  %3.10f \n", (1.0 * nnzb) / ((mb * 1.0) * (nb * 1.0)));  

    printf("cusparseSbsrmm...\n");

    float time;
    hipEvent_t start, stop;

    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    status = hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, dim, nb, nnzb, &fone, bsrDescr, bsrVal,
                            bsrRowPtr, bsrColInd, blockDim, y, nb * blockDim, &fzero, z, mb * blockDim);

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
                           
    printf("bsrmm cost time:  %3.10f ms \n", time);   

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("bsrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(mb * blockDim * dim * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed (z)");
        return 1;
    }

    status = hipsparseDestroyMatDescr(csrDescr);
    csrDescr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroyMatDescr(bsrDescr);
    bsrDescr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("BSR matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    CLEANUP("end");

    return 0;
}