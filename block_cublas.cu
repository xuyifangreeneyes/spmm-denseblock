#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include "hipblas.h"

template <typename T>
T* vec2ptr(std::vector<T> v) {
  T* ptr = (T*)malloc(v.size() * sizeof(T));
  for (int i = 0; i < v.size(); ++i) {
    ptr[i] = v[i];
  }
  return ptr;
}

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (bsrRowPtr) hipFree(bsrRowPtr);     \
    if (bsrColInd) hipFree(bsrColInd);     \
    if (bsrVal) hipFree(bsrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                                            \
  if (err != hipSuccess) {                                          \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP("");                                                     \
    exit(-1);                                                        \
  }

#define HANDLE_CUBLAS_ERROR(err)                                     \
  if (err != HIPBLAS_STATUS_SUCCESS) {                                \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP("");                                                     \
    exit(-1);                                                        \
  }

int main() {
  int m = 4;
  int n = 4;
  int dim = 3;
  int bsize = 2;
  int mb = (m + bsize - 1) / bsize;
  int nb = (n + bsize - 1) / bsize;
  assert(mb * bsize == m && nb * bsize == n);
  int nnzb = 3;
  float alpha = 1.0;
  float beta = 1.0;
  // float* hostAlpha = 0;
  // float* hostBeta = 0;
  // float* alpha = 0;
  // float* beta = 0;

  int* hostBsrRowPtr = 0;
  int* hostBsrColInd = 0;
  float* hostBsrVal = 0;
  int* bsrRowPtr = 0;
  int* bsrColInd = 0;
  float* bsrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  // hostAlpha = (float*) malloc(sizeof(float));
  // hostBeta = (float*) malloc(sizeof(float));
  // *hostAlpha = 1.0;
  // *hostBeta = 1.0;
  // HANDLE_ERROR( cudaMalloc((void**)&alpha, sizeof(float)) );
  // HANDLE_ERROR( cudaMalloc((void**)&beta, sizeof(float)) );
  // HANDLE_ERROR( cudaMemcpy(alpha, hostAlpha, (size_t) sizeof(float),
  // cudaMemcpyHostToDevice) );
  // HANDLE_ERROR( cudaMemcpy(beta, hostBeta, (size_t) sizeof(float),
  // cudaMemcpyHostToDevice) );

  hostBsrRowPtr = vec2ptr<int>({0, 2, 3});
  hostBsrColInd = vec2ptr<int>({0, 1, 1});
  hostBsrVal = vec2ptr<float>({0, 3, 1, 2, 4, 2, 0, 0, 0, 5, 0, 8});
  yHostPtr = vec2ptr<float>({6, 0, 0, 0, 7, 5, 4, 3, 0, 0, 0, 7});
  zHostPtr = (float*)malloc(m * dim * sizeof(float));

  HANDLE_ERROR(hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&bsrColInd, nnzb * sizeof(int)));
  HANDLE_ERROR(
      hipMalloc((void**)&bsrVal, nnzb * bsize * bsize * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(bsrRowPtr, hostBsrRowPtr,
                          (size_t)((mb + 1) * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(bsrColInd, hostBsrColInd,
                          (size_t)(nnzb * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(bsrVal, hostBsrVal,
                          (size_t)(nnzb * bsize * bsize * sizeof(float)),
                          hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMalloc((void**)&y, n * dim * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&z, m * dim * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset((void*)z, 0, m * dim * sizeof(float)));

  hipblasHandle_t handle;
  HANDLE_CUBLAS_ERROR(hipblasCreate(&handle));

  hipStream_t* streams = (hipStream_t*)malloc(mb * sizeof(hipStream_t));
  for (int i = 0; i < mb; ++i) {
    HANDLE_ERROR(hipStreamCreate(&streams[i]));
  }

  for (int i = 0; i < mb; ++i) {
    HANDLE_CUBLAS_ERROR(hipblasSetStream(handle, streams[i]));

    int start = hostBsrRowPtr[i], end = hostBsrRowPtr[i + 1];
    for (int j = start; j < end; ++j) {
      int idx = hostBsrColInd[j];
      float* A = bsrVal + j * bsize * bsize;
      float* B = y + idx * bsize * dim;
      float* C = z + i * bsize;
      HANDLE_CUBLAS_ERROR(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, bsize,
                                      dim, bsize, &alpha, A, bsize, B, dim,
                                      &beta, C, m));
    }
  }

  HANDLE_ERROR(hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)),
                          hipMemcpyDeviceToHost));

  for (int i = 0; i < m * dim; ++i) {
    printf("%f ", zHostPtr[i]);
  }
  printf("\n");

  CLEANUP("end");

  return 0;
}
