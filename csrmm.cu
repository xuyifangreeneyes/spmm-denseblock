#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr); \
    if (cooColIndexHostPtr) free(cooColIndexHostPtr); \
    if (cooValHostPtr) free(cooValHostPtr); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (cooRowIndex) hipFree(cooRowIndex); \
    if (cooColIndex) hipFree(cooColIndex); \
    if (cooVal) hipFree(cooVal); \
    if (descr) hipsparseDestroyMatDescr(descr); \
    if (handle) hipsparseDestroy(handle); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \


int main() {
    hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;
    int* cooRowIndexHostPtr = 0;
    int* cooColIndexHostPtr = 0;
    float* cooValHostPtr = 0;
    int* cooRowIndex = 0;
    int* cooColIndex = 0;
    float* cooVal = 0;
    int* csrRowPtr = 0;
    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;
    int n, nnz;
    float fzero = 0.0;
    float fone = 1.0;

    printf("testing examples\n");
    n = 4; nnz = 9;
    cooRowIndexHostPtr = (int*) malloc(nnz * sizeof(int));
    cooColIndexHostPtr = (int*) malloc(nnz * sizeof(int));
    cooValHostPtr = (float*) malloc(nnz * sizeof(float));
    if ((!cooRowIndexHostPtr) || (!cooColIndexHostPtr) || (!cooValHostPtr)) {
        CLEANUP("Host malloc failed (coo matrix)");
        return 1;
    }

    cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=1.0;
    cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=2; cooValHostPtr[1]=2.0;
    cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=3; cooValHostPtr[2]=3.0;
    cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=1; cooValHostPtr[3]=4.0;
    cooRowIndexHostPtr[4]=2; cooColIndexHostPtr[4]=0; cooValHostPtr[4]=5.0;
    cooRowIndexHostPtr[5]=2; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=6.0;
    cooRowIndexHostPtr[6]=2; cooColIndexHostPtr[6]=3; cooValHostPtr[6]=7.0;
    cooRowIndexHostPtr[7]=3; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=8.0;
    cooRowIndexHostPtr[8]=3; cooColIndexHostPtr[8]=3; cooValHostPtr[8]=9.0;

    yHostPtr = (float*) malloc(2 * n * sizeof(float));
    zHostPtr = (float*) malloc(2 * n * sizeof(float));
    if ((!yHostPtr) || (!zHostPtr)) {
        CLEANUP("Host malloc failed (dense matrix)");
        return 1;
    }

    yHostPtr[0] = 10.0;  
    yHostPtr[1] = 20.0;  
    yHostPtr[2] = 30.0;
    yHostPtr[3] = 40.0;
    yHostPtr[4] = 50.0;
    yHostPtr[5] = 60.0;
    yHostPtr[6] = 70.0;
    yHostPtr[7] = 80.0;

    cudaStat1 = hipMalloc((void**)&cooRowIndex, nnz * sizeof(int));
    cudaStat2 = hipMalloc((void**)&cooColIndex, nnz * sizeof(int));
    cudaStat3 = hipMalloc((void**)&cooVal, nnz * sizeof(float));
    cudaStat4 = hipMalloc((void**)&y, 2 * n * sizeof(float));

    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess || cudaStat4 != hipSuccess) {
        CLEANUP("Device malloc failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(cooColIndex, cooColIndexHostPtr, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice);
    cudaStat4 = hipMemcpy(y, yHostPtr, (size_t)(2 * n * sizeof(float)), hipMemcpyHostToDevice);

    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess || cudaStat4 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor initialization failed");
        return 1;
    }

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    cudaStat1 = hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(int));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed (csrRowPtr)");
        return 1;
    }

    status = hipsparseXcoo2csr(handle, cooRowIndex, nnz, n, csrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Conversion from COO to CSR format failed");
        return 1;
    }

    int devId;
    hipDeviceProp_t prop;
    hipError_t cudaStat;
    cudaStat = hipGetDevice(&devId);
    if (cudaStat != hipSuccess) {
        CLEANUP("hipGetDevice failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }
    cudaStat = hipGetDeviceProperties(&prop, devId);
    if (cudaStat != hipSuccess) {
        CLEANUP("hipGetDeviceProperties failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }

    cudaStat1 = hipMalloc((void**)&z, 2 * n * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed (z)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z, 0, 2 * n * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed");
        return 1;
    }

    status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, 2, n, nnz, 
                            &fone, descr, cooVal, csrRowPtr, cooColIndex, y, n, &fzero, z, n);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("csrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(2 * n * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed");
        return 1;
    }

    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    for (int i = 0; i < 2 * n; ++i) {
        printf("%f ", zHostPtr[i]);
    }

    printf("\n");
    CLEANUP("end");
}

