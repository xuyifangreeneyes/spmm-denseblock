#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <fstream>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "hipsparse.h"

static void handle_error(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(-1);
    }
}

#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

std::mt19937_64 gen(1234);

void readCSRMatrix(int m, int n, int nnz, int** hostCsrRowPtr, int** hostCsrColInd, float** hostCsrVal) {
    *hostCsrRowPtr = (int*) malloc((m + 1) * sizeof(int));
    *hostCsrColInd = (int*) malloc(nnz * sizeof(int));
    *hostCsrVal = (float*) malloc(nnz * sizeof(float));
    
    std::fstream s1("ddi_bfs_indptr.txt");
    std::fstream s2("ddi_bfs_indices.txt");
    int _m_1;
    s1 >> _m_1;
    assert(m + 1 == _m_1);
    for (int i = 0; i <= m; ++i) {
        int x;
        s1 >> x;
        (*hostCsrRowPtr)[i] = x;
    }

    int _nnz;
    s2 >> _nnz;
    assert(nnz == _nnz);
    for (int i = 0; i < nnz; ++i) {
        int x;
        s2 >> x;
        (*hostCsrColInd)[i] = x;
        (*hostCsrVal)[i] = 1.0;
    }
}

float* randomDenseMatrix(int n, int dim, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (csrColInd) hipFree(csrColInd); \
    if (csrVal) hipFree(csrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

int main() {
    hipError_t cudaStat1, cudaStat2, cudaStat3;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 4267;
    int n = m;    
    int nnz = 2135822;
    int dim = 64;
    float fzero = 0.0;
    float fone = 1.0;

    printf("density:  %3.10f ms \n", (1.0 * nnz) / ((m * 1.0) * (n * 1.0))); 

    int* hostCsrRowPtr = 0;
    int* hostCsrColInd = 0;
    float* hostCsrVal = 0;
    int* csrRowPtr = 0;
    int* csrColInd = 0;
    float* csrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    printf("read CSR matrix...\n");

    readCSRMatrix(m, n, nnz, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);

    printf("gpu memory malloc and memcpy...\n");

    cudaStat1 = hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&csrColInd, nnz * sizeof(int));
    cudaStat3 = hipMalloc((void**)&csrVal, nnz * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (CSR matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((m + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (CSR matrix)");
        return 1;
    }

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    cudaStat1 = hipMalloc((void**)&y, n * dim * sizeof(float));
    cudaStat2 = hipMalloc((void**)&z, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Device malloc failed (dense matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (y)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z, 0, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed (z)");
        return 1;
    }

    printf("hipsparseScsrmm...\n");

    float time;
    hipEvent_t start, stop;

    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, dim, n, nnz,
                            &fone, descr, csrVal, csrRowPtr, csrColInd, y, n, &fzero, z, m);

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );

    printf("csrmm cost time:  %3.10f ms \n", time);  
    
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("csrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed (z)");
        return 1;
    }

    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    CLEANUP("end");

    return 0;
}    