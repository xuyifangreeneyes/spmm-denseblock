#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <random>
#include <vector>
#include <string>
#include <iostream>
#include "hipsparse.h"
#include "load_data.h"
#include "utility.h"

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                     \
  if (!checkError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUDA ERROR");                    \
    exit(-1);                                 \
  }

#define HANDLE_CUSPARSE_ERROR(err)                    \
  if (!checkCusparseError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUSPARSE ERROR");                        \
    exit(-1);                                         \
  }

int main(int argc, char* argv[]) {
  std::string prefix = "tmp/" + std::string(argv[1]);
  std::cout << prefix << std::endl;
  int dim = std::stoi(argv[2]);

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;

  printf("load CSR matrix...\n");
  std::pair<int, int> pair = loadCSRFromFile(prefix, &hostCsrRowPtr, &hostCsrColInd);
  int n = pair.first;
  int nnz = pair.second;
  std::cout << "n=" << n << " nnz=" << nnz << std::endl;
  hostCsrVal = vec2ptr(std::vector<float>(nnz, 1.0));
  
  float alpha = 1.0;
  float beta = 1.0;

  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  printf("gpu memory malloc and memcpy...\n");

  HANDLE_ERROR( hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrColInd, nnz * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrVal, nnz * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((n + 1) * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_CUSPARSE_ERROR( hipsparseCreate(&handle) );

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&descr) );
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  yHostPtr = randomDenseMatrix(n, dim);
  zHostPtr = (float*)malloc(n * dim * sizeof(float));

  HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void**)&z, n * dim * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_ERROR( hipMemset((void*)z, 0, n * dim * sizeof(float)) );

  printf("hipsparseScsrmm...\n");

  float time;
  hipEvent_t start, stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  HANDLE_CUSPARSE_ERROR( hipsparseScsrmm2(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_TRANSPOSE, n, dim, n, nnz, &alpha,
      descr, csrVal, csrRowPtr, csrColInd, y, dim, &beta, z, n) );

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

  printf("csrmm cost time:  %3.10f ms \n", time);

  HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(n * dim * sizeof(float)),
                           hipMemcpyDeviceToHost) );

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(descr));
  descr = 0;
  HANDLE_CUSPARSE_ERROR(hipsparseDestroy(handle));
  handle = 0;

  CLEANUP("end");

  return 0;
}