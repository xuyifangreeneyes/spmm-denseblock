#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <random>
#include <vector>
#include <string>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "hipsparse.h"
#include "load_data.h"
#include "gespmm_csrmm.h"
#include "utility.h"

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                     \
  if (!checkError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUDA ERROR");                    \
    exit(-1);                                 \
  }

#define HANDLE_CUSPARSE_ERROR(err)                    \
  if (!checkCusparseError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUSPARSE ERROR");                        \
    exit(-1);                                         \
  }

int main(int argc, char* argv[]) {
  std::string prefix = "tmp/" + std::string(argv[1]);
  int dim = std::stoi(argv[2]);
  std::string csrmmImpl(argv[3]);
  int transposeB = std::stoi(argv[4]);
  printf("graph = %s dim = %d csrmmImpl = %s transposeB = %d\n", 
         argv[1], dim, csrmmImpl.c_str(), transposeB);

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;

  printf("load CSR matrix...\n");
  std::pair<int, int> pair = loadCSRFromFile(prefix, &hostCsrRowPtr, &hostCsrColInd);
  int n = pair.first;
  int nnz = pair.second;
  std::cout << "n=" << n << " nnz=" << nnz << std::endl;
  hostCsrVal = vec2ptr(std::vector<float>(nnz, 1.0));
  
  float alpha = 1.0;
  float beta = 0.0;

  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  hipsparseOperation_t transB;
  int ldb;
  if (transposeB == 0) {
    transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    ldb = n;
  } else if (transposeB == 1) {
    transB = HIPSPARSE_OPERATION_TRANSPOSE;
    ldb = dim;
  } else {
    assert(false);
  }

  printf("gpu memory malloc and memcpy...\n");

  HANDLE_ERROR( hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrColInd, nnz * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrVal, nnz * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((n + 1) * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_CUSPARSE_ERROR( hipsparseCreate(&handle) );

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&descr) );
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  yHostPtr = randomDenseMatrix(n, dim);
  zHostPtr = (float*)malloc(n * dim * sizeof(float));

  HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void**)&z, n * dim * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_ERROR( hipMemset((void*)z, 0, n * dim * sizeof(float)) );

  printf("hipsparseScsrmm...\n");

  int epoch = 10;
  float totalTime = 0;

  for (int i = 0; i < epoch; ++i) {
    float time;
    hipEvent_t start, stop;

    // hipProfilerStart();

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    if (csrmmImpl == "hipsparseScsrmm") {
      assert(transB == HIPSPARSE_OPERATION_NON_TRANSPOSE);
      HANDLE_CUSPARSE_ERROR( hipsparseScsrmm(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, dim, n, nnz, &alpha,
        descr, csrVal, csrRowPtr, csrColInd, y, ldb, &beta, z, n) );
    } else if (csrmmImpl == "hipsparseScsrmm2") {
      HANDLE_CUSPARSE_ERROR( hipsparseScsrmm2(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        transB, n, dim, n, nnz, &alpha,
        descr, csrVal, csrRowPtr, csrColInd, y, ldb, &beta, z, n) );    
    } else if (csrmmImpl == "gespmm") {
      gespmm_csrmm<float>(n, dim, csrRowPtr, csrColInd, csrVal, y, z);
    } else {
      assert(false);
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

    // hipProfilerStop();

    printf("csrmm cost time:  %3.10f ms \n", time);
    totalTime += time;
  }

  printf("average csrmm cost time: %3.10f ms\n", totalTime / epoch);

  HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(n * dim * sizeof(float)),
                           hipMemcpyDeviceToHost) );

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(descr));
  descr = 0;
  HANDLE_CUSPARSE_ERROR(hipsparseDestroy(handle));
  handle = 0;

  CLEANUP("end");

  return 0;
}