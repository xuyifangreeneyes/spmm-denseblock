#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <random>
#include <fstream>
#include <string>
#include <assert.h>
#include <thread>
#include <algorithm>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

std::mt19937_64 gen(1234);

template<typename T>
T* vec2ptr(std::vector<T> v) {
    T* ptr = (T*) malloc(v.size() * sizeof(T));
    for (int i = 0; i < v.size(); ++i) {
        ptr[i] = v[i];
    }
    return ptr;
}

int randomBSRMatrix(int mb, int nb, int bsize, float p, int** hostBsrRowPtr, int** hostBsrColInd, float** hostBsrVal, float minVal=-1, float maxVal=1) {
    std::uniform_real_distribution<float> flip(0, 1), dist(minVal, maxVal);
    int blockNum = bsize * bsize;
    *hostBsrRowPtr = (int*) malloc((mb + 1) * sizeof(int));
    int cnt = 0;
    (*hostBsrRowPtr)[0] = cnt;
    std::vector<int> indices;
    std::vector<float> vals;
    for (int i = 1; i <= mb; ++i) {
        for (int j = 0; j < nb; ++j) {
            if (flip(gen) < p) {
                indices.push_back(j);
                for (int k = 0; k < blockNum; ++k) {
                    vals.push_back(dist(gen));
                }
                ++cnt;
            }
        }
        (*hostBsrRowPtr)[i] = cnt;
        // if (i % 1000 == 0) {
        //     printf("i = %d\n", i);
        // }
    }
    *hostBsrColInd = vec2ptr(std::move(indices));
    *hostBsrVal = vec2ptr(std::move(vals));

    // Generating random BSR matrix may be time-consuming, so we record it for next time use.
    // std::string bd = std::to_string(bsize);
    // std::fstream s1("bsr_" + bd + "_indptr.txt", std::ios::out | std::ios::trunc);
    // std::fstream s2("bsr_" + bd + "_indices.txt", std::ios::out | std::ios::trunc);

    // s1 << mb + 1 << std::endl;
    // for (int i = 0; i <= mb; ++i) {
    //     s1 << (*hostBsrRowPtr)[i] << " ";
    // }
    // s1 << std::endl;

    // s2 << cnt << std::endl;
    // for (int i = 0; i < cnt; ++i) {
    //     s2 << (*hostBsrColInd)[i] << " ";
    // }
    // s2 << std::endl;

    return cnt;
}

int readAndFillBSRMatrix(int mb, int nb, int bsize, int** hostBsrRowPtr, int** hostBsrColInd, float** hostBsrVal, float minVal=-1, float maxVal=1) {
    std::string bd = std::to_string(bsize);
    std::fstream s1("bsr_" + bd + "_indptr.txt", std::ios::in);
    std::fstream s2("bsr_" + bd + "_indices.txt", std::ios::in);

    int xx;
    s1 >> xx;
    assert(mb + 1 == xx);
    *hostBsrRowPtr = (int*) malloc((mb + 1) * sizeof(int));
    for (int i = 0; i <= mb; ++i) {
        s1 >> (*hostBsrRowPtr)[i];
    }

    int nnzb;
    s2 >> nnzb;
    *hostBsrColInd = (int*) malloc(nnzb * sizeof(int));
    for (int i = 0; i < nnzb; ++i) {
        s2 >> (*hostBsrColInd)[i];
    }

    int num = nnzb * bsize * bsize;
    *hostBsrVal = (float*) malloc(num * sizeof(float));
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    for (int i = 0; i < num; ++i) {
        (*hostBsrVal)[i] = dist(gen);
    }

    return nnzb;
}    

float* randomDenseMatrix(int n, int dim, float minVal=-1, float maxVal=1) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0)

#define HANDLE_ERROR( err ) \
if (err != hipSuccess) { \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP(""); \
    exit(-1); \
}

#define HANDLE_CUBLAS_ERROR( err ) \
if (err != HIPBLAS_STATUS_SUCCESS) { \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP(""); \
    exit(-1); \
}

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int bsize = std::stoi(argv[2]); 
    int dim = std::stoi(argv[3]);
    printf("p = %f bsize = %d dim = %d\n", p, bsize, dim);

    int m = 2 << 16;
    int n = m;
    int mb = (m + bsize - 1) / bsize;
    int nb = (n + bsize - 1) / bsize;
    assert(mb * bsize == m && nb * bsize == n);
    int nnzb = 0;
    float alpha = 1.0;
    float beta = 1.0;

    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    printf("generate random BSR matrix\n");

    // nnzb = randomBSRMatrix(mb, nb, bsize, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);
    nnzb = readAndFillBSRMatrix(mb, nb, bsize, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);

    printf("nnzb = %d, density of BSR matrix is %f\n", nnzb, (nnzb * 1.0) / mb / nb);

    printf("gpu memory malloc and memcpy...\n");

    HANDLE_ERROR( hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrColInd, nnzb * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrVal, nnzb * bsize * bsize * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * bsize * bsize * sizeof(float)), hipMemcpyHostToDevice) );

    printf("prepare y and z...\n");

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**)&z, m * dim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemset((void*)z, 0, m * dim * sizeof(float)) );

    hipblasHandle_t handle;
    HANDLE_CUBLAS_ERROR( hipblasCreate(&handle) );

    printf("block cublas...\n");

    hipStream_t *streams = (hipStream_t*)malloc(mb * sizeof(hipStream_t));
    for (int i = 0; i < mb; ++i) {
        HANDLE_ERROR( hipStreamCreate(&streams[i]) );
    }

    float time;
    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    // auto streamJob = [=](int i) {
    //     HANDLE_CUBLAS_ERROR( cublasSetStream(handle, streams[i]) );

    //     int start = hostBsrRowPtr[i], end = hostBsrRowPtr[i + 1];
    //     for (int j = start; j < end; ++j) {
    //         int idx = hostBsrColInd[j];
    //         float* A = bsrVal + j * bsize * bsize;
    //         float* B = y + idx * bsize * dim;
    //         float* C = z + i * bsize;
    //         HANDLE_CUBLAS_ERROR( cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_T, 
    //                                          bsize, dim, bsize, &alpha, 
    //                                          A, bsize, B, dim, &beta, 
    //                                          C, m) );
    //     }
    // };

    // std::vector<std::thread> threads;
    // for (int i = 0; i < mb; ++i) {
    //     threads.emplace_back(streamJob, i);
    // }

    // for (auto&& t : threads) {
    //     t.join();
    // }

    for (int i = 0; i < mb; ++i) {
        HANDLE_CUBLAS_ERROR( hipblasSetStream(handle, streams[i]) );

        int start = hostBsrRowPtr[i], end = hostBsrRowPtr[i + 1];
        for (int j = start; j < end; ++j) {
            int idx = hostBsrColInd[j];
            float* A = bsrVal + j * bsize * bsize;
            float* B = y + idx * bsize * dim;
            float* C = z + i * bsize;
            HANDLE_CUBLAS_ERROR( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                                             bsize, dim, bsize, &alpha, 
                                             A, bsize, B, dim, &beta, 
                                             C, m) );
        }
    }

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    printf("bsrmm cost time:  %3.10f ms \n", time);   

    HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost) );

    for (int i = 0; i < mb; ++i) {
        HANDLE_ERROR( hipStreamDestroy(streams[i]) );
    }

    CLEANUP("end");

    return 0;
}    