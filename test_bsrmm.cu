#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <vector>
#include <random>
#include <fstream>
#include <string>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "hip/hip_runtime_api.h"
#include "load_data.h"
#include "utility.h"
#include "rocsparse_bsrmm.h"


#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0)

#define HANDLE_ERROR( err ) \
if (!checkError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUDA ERROR"); \
    exit(-1); \
}

#define HANDLE_CUSPARSE_ERROR( err ) \
if (!checkCusparseError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUSPARSE ERROR"); \
    exit(-1); \
}

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int blockDim = std::stoi(argv[2]); 
    int dim = std::stoi(argv[3]);
    std::string bsrmmImpl(argv[4]);
    int transposeB = std::stoi(argv[5]);
    printf("p = %f blockDim = %d dim = %d bsrmmImpl = %s transposeB = %d\n", p, blockDim, dim, bsrmmImpl.c_str(), transposeB);

    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 2 << 16;
    int n = m;
    int mb = (m + blockDim - 1) / blockDim;
    int nb = (n + blockDim - 1) / blockDim;
    assert(mb * blockDim == m && nb * blockDim == n);
    int nnzb = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    hipsparseOperation_t transB;
    int ldb;
    if (transposeB == 0) {
        transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        ldb = n;
    } else if (transposeB == 1) {
        transB = HIPSPARSE_OPERATION_TRANSPOSE;
        ldb = dim;
    } else {
        assert(false);
    }

    printf("generate random BSR matrix\n");

    // nnzb = randomBSRMatrix(mb, nb, blockDim, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal, -1, 1, true);
    nnzb = readAndFillBSRMatrix(mb, nb, blockDim, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);

    printf("nnzb = %d mb = %d nb = %d\n", nnzb, mb, nb);
    printf("density of BSR matrix is %f\n", (nnzb * 1.0) / ((mb * 1.0) * (nb * 1.0)));

    printf("gpu memory malloc and memcpy...\n");

    HANDLE_ERROR( hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrColInd, nnzb * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * blockDim * blockDim * sizeof(float)), hipMemcpyHostToDevice) );
    
    HANDLE_CUSPARSE_ERROR( hipsparseCreate(&handle) );
    
    HANDLE_CUSPARSE_ERROR( hipsparseCreateMatDescr(&descr) );
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    printf("prepare y and z...\n");

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**)&z, m * dim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );

    HANDLE_ERROR( hipMemset((void*)z, 0, m * dim * sizeof(float)) );

    printf("hipsparseSbsrmm...\n"); 

    float time;
    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    hipProfilerStart();

    if (bsrmmImpl == "rocsparse") {
        HANDLE_CUSPARSE_ERROR( rocsparse_bsrmm_template<float>(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                               transB, mb, dim, nb, nnzb, fone, descr, bsrVal,
                                                               bsrRowPtr, bsrColInd, blockDim, y, ldb, fzero, z, m) ); 
    }  else if (bsrmmImpl == "cusparse"){
        HANDLE_CUSPARSE_ERROR( hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                              transB, mb, dim, nb, nnzb, &fone, descr, bsrVal,
                                              bsrRowPtr, bsrColInd, blockDim, y, ldb, &fzero, z, m) ); 
    } else {
        assert(false);
    }

    hipProfilerStop();

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    
    float gflops = (nnzb / 1.0e6) * (blockDim * blockDim * dim) / time; 
    printf("bsrmm cost time: %6.10f ms\nGFLOPs: %6.10f\n", time, gflops);   

    HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost) );

    HANDLE_CUSPARSE_ERROR( hipsparseDestroyMatDescr(descr) );
    descr = 0;
    HANDLE_CUSPARSE_ERROR( hipsparseDestroy(handle) );
    handle = 0;

    CLEANUP("end");

    return 0;
}