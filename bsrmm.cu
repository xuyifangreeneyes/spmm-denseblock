#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "hipsparse.h"

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (descr) hipsparseDestroyMatDescr(descr); \
    if (handle) hipsparseDestroy(handle); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

template<typename T>
T* vec2ptr(std::vector<T> v) {
    T* ptr = (T*) malloc(v.size() * sizeof(T));
    for (size_t i = 0; i < v.size(); ++i) {
        ptr[i] = v[i];
    }
    return ptr;
}

int main() {
    hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;
    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;
    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;
    int blockDim = 2;
    int mb = 2, kb = 3;
    int m = mb * blockDim, k = kb * blockDim;
    int n = 2;
    int nnzb = 4; 
    float fzero = 0.0;
    float fone = 1.0;

    hostBsrRowPtr = vec2ptr<int>({0, 2, 4});
    hostBsrColInd = vec2ptr<int>({0, 2, 1, 2});
    hostBsrVal = vec2ptr<float>({0, 4, 2, 7, 1, 8, 2, 0, 9, 0, 0, 2, 0, 6, 7, 0});
    yHostPtr = vec2ptr<float>({1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12});
    zHostPtr = (float*) malloc(m * n * sizeof(float));

    if ((!hostBsrRowPtr) || (!hostBsrColInd) || (!hostBsrVal) || (!yHostPtr) || (!zHostPtr)) {
        CLEANUP("Host malloc failed");
        return 1;
    }

    cudaStat1 = hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&bsrColInd, nnzb * sizeof(int));
    cudaStat3 = hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float));
    cudaStat4 = hipMalloc((void**)&y, k * n * sizeof(float));

    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess || cudaStat4 != hipSuccess) {
        CLEANUP("Device malloc failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * blockDim * blockDim * sizeof(float)), hipMemcpyHostToDevice);
    cudaStat4 = hipMemcpy(y, yHostPtr, (size_t)(k * n * sizeof(float)), hipMemcpyHostToDevice);

    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess || cudaStat4 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed");
        return 1;
    }

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }

    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor initialization failed");
        return 1;
    }

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int devId;
    hipDeviceProp_t prop;
    hipError_t cudaStat;
    cudaStat = hipGetDevice(&devId);
    if (cudaStat != hipSuccess) {
        CLEANUP("hipGetDevice failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }
    cudaStat = hipGetDeviceProperties(&prop, devId);
    if (cudaStat != hipSuccess) {
        CLEANUP("hipGetDeviceProperties failed");
        printf("Error: cudaStat %d, %s\n", cudaStat, hipGetErrorString(cudaStat));
        return 1;
    }

    cudaStat1 = hipMalloc((void**)&z, m * n * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Device malloc failed (z)");
        return 1;
    }
    cudaStat1 = hipMemset((void*)z, 0, m * n * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed");
        return 1;
    }

    status = hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, n, kb, nnzb, &fone, descr, bsrVal,
                            bsrRowPtr, bsrColInd, blockDim, y, k, &fzero, z, m);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("csrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(m * n * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed");
        return 1;
    }

    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    for (int i = 0; i < m * n; ++i) {
        printf("%f ", zHostPtr[i]);
    }

    printf("\n");
    CLEANUP("end");
}