#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <random>
#include <vector>
#include <iostream>
#include "rocsparse_bsrmm.h"
#include "hipsparse.h"
#include "load_data.h"
#include "utility.h"

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (bsrRowPtr) hipFree(bsrRowPtr);     \
    if (bsrColInd) hipFree(bsrColInd);     \
    if (bsrVal) hipFree(bsrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                     \
  if (!checkError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUDA ERROR");                    \
    exit(-1);                                 \
  }

#define HANDLE_CUSPARSE_ERROR(err)                    \
  if (!checkCusparseError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUSPARSE ERROR");                        \
    exit(-1);                                         \
  }

int main(int argc, char* argv[]) {
  std::string prefix = "tmp/" + std::string(argv[1]);
  std::cout << prefix << std::endl;
  int blockDim = std::stoi(argv[2]);
  int dim = std::stoi(argv[3]);
  std::string bsrmmImpl(argv[4]);

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t csrDescr = 0, bsrDescr = 0;

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;

  printf("load CSR matrix...\n");
  std::pair<int, int> pair = loadCSRFromFile(prefix, &hostCsrRowPtr, &hostCsrColInd);
  int n = pair.first;
  int nnz = pair.second;
  std::cout << "n=" << n << " nnz=" << nnz << std::endl;
  hostCsrVal = vec2ptr(std::vector<float>(nnz, 1.0));

  int nb = (n + blockDim - 1) / blockDim;
  int nnzb = 0;
  float alpha = 1.0;
  float beta = 1.0;

  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  int* bsrRowPtr = 0;
  int* bsrColInd = 0;
  float* bsrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  yHostPtr = randomDenseMatrix(nb * blockDim, dim);
  zHostPtr = (float*)malloc(nb * blockDim * dim * sizeof(float));

  HANDLE_ERROR( hipMalloc((void**)&y, nb * blockDim * dim * sizeof(float)) );
  HANDLE_ERROR( hipMalloc((void**)&z, nb * blockDim * dim * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(nb * blockDim * dim * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_ERROR( hipMemset((void*)z, 0, nb * blockDim * dim * sizeof(float)) );

  printf("gpu memory malloc and memcpy...\n");

  HANDLE_ERROR( hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrColInd, nnz * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&csrVal, nnz * sizeof(float)) );

  HANDLE_ERROR( hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((n + 1) * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)), hipMemcpyHostToDevice) );

  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&csrDescr));
  hipsparseSetMatType(csrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(csrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&bsrDescr));
  hipsparseSetMatType(bsrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(bsrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  HANDLE_ERROR( hipMalloc((void**)&bsrRowPtr, (nb + 1) * sizeof(int)) );

  int base;
  int* nnzTotalDevHostPtr = &nnzb;

  HANDLE_CUSPARSE_ERROR( hipsparseXcsr2bsrNnz(
      handle, HIPSPARSE_DIRECTION_ROW, n, n, csrDescr,
      csrRowPtr, csrColInd, blockDim, bsrDescr,
      bsrRowPtr, nnzTotalDevHostPtr) );
  if (NULL != nnzTotalDevHostPtr) {
    nnzb = *nnzTotalDevHostPtr;
  } else {
    HANDLE_ERROR( hipMemcpy(&nnzb, bsrRowPtr + nb, sizeof(int), hipMemcpyDeviceToHost) );
    HANDLE_ERROR( hipMemcpy(&base, bsrRowPtr, sizeof(int), hipMemcpyDeviceToHost) );
    nnzb -= base;
  }

  long long numVal =
      (long long)nnzb * (long long)(blockDim * blockDim) * sizeof(float);
  printf("numVal = %lld\n", numVal);
  HANDLE_ERROR( hipMalloc((void**)&bsrColInd, nnzb * sizeof(int)) );
  HANDLE_ERROR( hipMalloc((void**)&bsrVal, numVal) );

  HANDLE_CUSPARSE_ERROR( hipsparseScsr2bsr(
      handle, HIPSPARSE_DIRECTION_ROW, n, n, csrDescr,
      csrVal, csrRowPtr, csrColInd, blockDim, bsrDescr,
      bsrVal, bsrRowPtr, bsrColInd) );

  printf("density:  %3.10f \n", (1.0 * nnzb) / ((nb * 1.0) * (nb * 1.0)));

  printf("hipsparseSbsrmm...\n");

  float time;
  hipEvent_t start, stop;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  if (bsrmmImpl == "rocsparse") {
    HANDLE_CUSPARSE_ERROR( rocsparse_bsrmm_template<float>(
        handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, nb, dim, nb, nnzb, alpha, bsrDescr, bsrVal,
        bsrRowPtr, bsrColInd, blockDim, y, nb * blockDim, beta, z, nb * blockDim) );
  } else if (bsrmmImpl == "cusparse") {
    HANDLE_CUSPARSE_ERROR( hipsparseSbsrmm(
        handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, nb, dim, nb, nnzb, &alpha, bsrDescr, bsrVal,
        bsrRowPtr, bsrColInd, blockDim, y, nb * blockDim, &beta, z, nb * blockDim) );
  }

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

  printf("bsrmm cost time:  %3.10f ms \n", time);

  HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(nb * blockDim * dim * sizeof(float)), hipMemcpyDeviceToHost) );

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(csrDescr));
  csrDescr = 0;

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(bsrDescr));
  bsrDescr = 0;

  HANDLE_CUSPARSE_ERROR(hipsparseDestroy(handle));
  handle = 0;

  CLEANUP("end");

  return 0;
}