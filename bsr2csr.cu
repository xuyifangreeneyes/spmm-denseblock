#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include "hipsparse.h"

static void handle_error(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(-1);
    }
}

#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

std::mt19937_64 gen(1234);

template<typename T>
T* vec2ptr(std::vector<T> v) {
    T* ptr = (T*) malloc(v.size() * sizeof(T));
    for (size_t i = 0; i < v.size(); ++i) {
        ptr[i] = v[i];
    }
    return ptr;
}

int randomBSRMatrix(int mb, int nb, int blockDim, float p, int** hostBsrRowPtr, int** hostBsrColInd, float** hostBsrVal, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> flip(0, 1), dist(minVal, maxVal);
    int blockNum = blockDim * blockDim;
    *hostBsrRowPtr = (int*) malloc((mb + 1) * sizeof(int));
    int cnt = 0;
    (*hostBsrRowPtr)[0] = cnt;
    std::vector<int> indices;
    std::vector<float> vals;
    for (int i = 1; i <= mb; ++i) {
        for (int j = 0; j < nb; ++j) {
            if (flip(gen) < p) {
                indices.push_back(j);
                for (int k = 0; k < blockNum; ++k) {
                    vals.push_back(dist(gen));
                }
                ++cnt;
            }
        }
        (*hostBsrRowPtr)[i] = cnt;
    }
    *hostBsrColInd = vec2ptr(std::move(indices));
    *hostBsrVal = vec2ptr(std::move(vals));

    return cnt;
}

float* randomDenseMatrix(int n, int dim, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (z1HostPtr) free(z1HostPtr); \
    if (z2HostPtr) free(z2HostPtr); \
    if (csrRowPtr) hipFree(csrRowPtr); \
    if (csrColInd) hipFree(csrColInd); \
    if (csrVal) hipFree(csrVal); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (z1) hipFree(z1); \
    if (z2) hipFree(z2); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int blockDim = std::stof(argv[2]); 
    printf("p = %f blockDim = %d\n", p, blockDim);

    hipError_t cudaStat1, cudaStat2, cudaStat3;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t csrDescr = 0, bsrDescr = 0;

    int m = 4096;
    int n = 4096;
    int mb = m / blockDim;
    int nb = n / blockDim;
    int dim = 100;
    int nnzb = 0;
    int nnz = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    int* csrRowPtr = 0;
    int* csrColInd = 0;
    float* csrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* z1HostPtr = 0;
    float* z1 = 0;
    float* z2HostPtr = 0;
    float* z2 = 0;

    nnzb = randomBSRMatrix(mb, nb, blockDim, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);

    cudaStat1 = hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&bsrColInd, nnzb * sizeof(int));
    cudaStat3 = hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (BSR matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * blockDim * blockDim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (BSR matrix)");
        return 1;
    }

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }
    
    status = hipsparseCreateMatDescr(&bsrDescr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("BSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(bsrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(bsrDescr, HIPSPARSE_INDEX_BASE_ZERO);

    status = hipsparseCreateMatDescr(&csrDescr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(csrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(csrDescr, HIPSPARSE_INDEX_BASE_ZERO);

    nnz = nnzb * blockDim * blockDim;
    cudaStat1 = hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&csrColInd, nnz * sizeof(int));
    cudaStat3 = hipMalloc((void**)&csrVal, nnz * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (CSR matrix)");
        return 1;
    }
    status = hipsparseSbsr2csr(handle, HIPSPARSE_DIRECTION_ROW, mb, nb, bsrDescr, bsrVal, bsrRowPtr, bsrColInd,
                              blockDim, csrDescr, csrVal, csrRowPtr, csrColInd);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("hipsparseSbsr2csr failed");
        return 1;
    }

    yHostPtr = randomDenseMatrix(n, dim);
    z1HostPtr = (float*) malloc(m * dim * sizeof(float));
    z2HostPtr = (float*) malloc(m * dim * sizeof(float));

    cudaStat1 = hipMalloc((void**)&y, n * dim * sizeof(float));
    cudaStat2 = hipMalloc((void**)&z1, m * dim * sizeof(float));
    cudaStat3 = hipMalloc((void**)&z2, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (dense matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (y)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z1, 0, m * dim * sizeof(float));
    cudaStat2 = hipMemset((void*)z2, 0, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Memset on Device failed (z)");
        return 1;
    }

    float time1, time2;
    hipEvent_t start1, stop1, start2, stop2;

    HANDLE_ERROR( hipEventCreate(&start2) );
    HANDLE_ERROR( hipEventCreate(&stop2) );
    HANDLE_ERROR( hipEventRecord(start2, 0) );

    status = hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, dim, nb, nnzb, &fone, bsrDescr, bsrVal,
                            bsrRowPtr, bsrColInd, blockDim, y, n, &fzero, z2, m);
    
    HANDLE_ERROR( hipEventRecord(stop2, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop2) );
    HANDLE_ERROR( hipEventElapsedTime(&time2, start2, stop2) );
                                               
    printf("bsrmm cost time:  %3.10f ms \n", time2);   

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("bsrmm failed");
        return 1;
    }

    HANDLE_ERROR( hipEventCreate(&start1) );
    HANDLE_ERROR( hipEventCreate(&stop1) );
    HANDLE_ERROR( hipEventRecord(start1, 0) );

    status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, dim, n, nnz,
                            &fone, csrDescr, csrVal, csrRowPtr, csrColInd, y, n, &fzero, z1, m);
    
    HANDLE_ERROR( hipEventRecord(stop1, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop1) );
    HANDLE_ERROR( hipEventElapsedTime(&time1, start1, stop1) );
                                               
    printf("csrmm cost time:  %3.10f ms \n", time1);   

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("csrmm failed");
        return 1;
    }

    float ratio = time2 / time1;
    printf("bsrmm time / csrmm time: %3.10f\n", ratio);

    cudaStat1 = hipMemcpy(z1HostPtr, z1, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(z2HostPtr, z2, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed (z)");
        return 1;
    }

    status = hipsparseDestroyMatDescr(csrDescr);
    csrDescr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CSR matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroyMatDescr(bsrDescr);
    bsrDescr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("BSR matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    bool flag = true;
    for (int i = 0; i < m * dim; ++i) {
        float error = fabs(z1HostPtr[i] - z2HostPtr[i]);
        if (error > 0.05) {
            printf("inconsistent result: %d %f", i, error);
            flag = false;
            break;
        }
    }

    if (flag) {
        printf("\nsame result\n");
    } else {
        printf("\ninconsistent result\n");
    }
    CLEANUP("end");

}