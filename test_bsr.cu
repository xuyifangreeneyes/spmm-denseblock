#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include "hipsparse.h"

static void handleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(-1);
    }
}

#define HANDLE_ERROR( err ) (handleError( err, __FILE__, __LINE__ ))

std::mt19937_64 gen(1234);

template<typename T>
T* vec2ptr(std::vector<T> v) {
    T* ptr = (T*) malloc(v.size() * sizeof(T));
    for (int i = 0; i < v.size(); ++i) {
        ptr[i] = v[i];
    }
    return ptr;
}

int randomBSRMatrix(int mb, int nb, int blockDim, float p, int** hostBsrRowPtr, int** hostBsrColInd, float** hostBsrVal, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> flip(0, 1), dist(minVal, maxVal);
    int blockNum = blockDim * blockDim;
    *hostBsrRowPtr = (int*) malloc((mb + 1) * sizeof(int));
    int cnt = 0;
    (*hostBsrRowPtr)[0] = cnt;
    std::vector<int> indices;
    std::vector<float> vals;
    for (int i = 1; i <= mb; ++i) {
        for (int j = 0; j < nb; ++j) {
            if (flip(gen) < p) {
                indices.push_back(j);
                for (int k = 0; k < blockNum; ++k) {
                    vals.push_back(dist(gen));
                }
                ++cnt;
            }
        }
        (*hostBsrRowPtr)[i] = cnt;
    }
    *hostBsrColInd = vec2ptr(std::move(indices));
    *hostBsrVal = vec2ptr(std::move(vals));

    return cnt;
}

float* randomDenseMatrix(int n, int dim, float minVal=-10, float maxVal=10) {
    std::uniform_real_distribution<float> dist(minVal, maxVal);
    int sz = n * dim;
    float* ptr = (float*) malloc(sz * sizeof(float));
    for (int i = 0; i < sz; ++i) {
        ptr[i] = dist(gen);
    }
    return ptr;
}

#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (z) hipFree(z); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0) \

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int blockDim = std::stoi(argv[2]); 
    int dim = std::stoi(argv[3]);
    printf("p = %f blockDim = %d dim = %d\n", p, blockDim, dim);

    hipError_t cudaStat1, cudaStat2, cudaStat3;
    hipsparseStatus_t status;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 131072;
    int n = m;
    int mb = m / blockDim;
    int nb = n / blockDim;
    int nnzb = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* zHostPtr = 0;
    float* z = 0;

    printf("generate random BSR matrix\n");

    nnzb = randomBSRMatrix(mb, nb, blockDim, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);

    printf("gpu memory malloc and memcpy...\n");

    cudaStat1 = hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int));
    cudaStat2 = hipMalloc((void**)&bsrColInd, nnzb * sizeof(int));
    cudaStat3 = hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Device malloc failed (BSR matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice);
    cudaStat3 = hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * blockDim * blockDim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess || cudaStat3 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (BSR matrix)");
        return 1;
    }
    
    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }
    
    status = hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("BSR Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    cudaStat1 = hipMalloc((void**)&y, n * dim * sizeof(float));
    cudaStat2 = hipMalloc((void**)&z, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
        CLEANUP("Device malloc failed (dense matrix)");
        return 1;
    }

    cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Host to Device failed (y)");
        return 1;
    }

    cudaStat1 = hipMemset((void*)z, 0, m * dim * sizeof(float));
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memset on Device failed (z)");
        return 1;
    }

    printf("cusparseSbsrmm...\n");

    float time;
    hipEvent_t start, stop;

    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    status = hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, dim, nb, nnzb, &fone, descr, bsrVal,
                            bsrRowPtr, bsrColInd, blockDim, y, n, &fzero, z, m);

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
                           
    printf("bsrmm cost time:  %3.10f ms \n", time);   

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("bsrmm failed");
        return 1;
    }

    cudaStat1 = hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost);
    if (cudaStat1 != hipSuccess) {
        CLEANUP("Memcpy from Device to Host failed (z)");
        return 1;
    }

    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }

    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }

    CLEANUP("end");

    return 0;
}