#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include <random>
#include <fstream>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "load_matrix.h"
// #include "hip/hip_runtime_api.h"


#define CLEANUP(s) \
do { \
    printf("%s\n", s); \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal); \
    if (yHostPtr) free(yHostPtr); \
    if (zHostPtr) free(zHostPtr); \
    if (bsrRowPtr) hipFree(bsrRowPtr); \
    if (bsrColInd) hipFree(bsrColInd); \
    if (bsrVal) hipFree(bsrVal); \
    if (y) hipFree(y); \
    if (y0) hipFree(y0); \
    if (z) hipFree(z); \
    if (z0) hipFree(z0); \
    hipDeviceReset(); \
    fflush(stdout); \
} while (0)

#define HANDLE_ERROR( err ) \
if (err != hipSuccess) { \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP(hipGetErrorString(err)); \
    exit(-1); \
}

#define HANDLE_CUSPARSE_ERROR( err, s ) \
if (err != HIPSPARSE_STATUS_SUCCESS) { \
    CLEANUP(s); \
    exit(-1); \
}

int main(int argc, char* argv[]) {
    float p = std::stof(argv[1]);
    int blockDim = std::stoi(argv[2]); 
    int dim = std::stoi(argv[3]);
    printf("p = %f blockDim = %d dim = %d\n", p, blockDim, dim);

    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;

    int m = 131072;
    int n = m;
    int mb = (m + blockDim - 1) / blockDim;
    int nb = (n + blockDim - 1) / blockDim;
    assert(mb * blockDim == m && nb * blockDim == n);
    int nnzb = 0;
    float fzero = 0.0;
    float fone = 1.0;

    int* hostBsrRowPtr = 0;
    int* hostBsrColInd = 0;
    float* hostBsrVal = 0;
    int* bsrRowPtr = 0;
    int* bsrColInd = 0;
    float* bsrVal = 0;

    float* yHostPtr = 0;
    float* y = 0;
    float* y0 = 0;
    float* zHostPtr = 0;
    float* z = 0;
    float* z0 = 0;

    printf("generate random BSR matrix\n");

    nnzb = randomBSRMatrix(mb, nb, blockDim, p, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);
    // nnzb = readAndFillBSRMatrix(mb, nb, blockDim, &hostBsrRowPtr, &hostBsrColInd, &hostBsrVal);

    printf("density of BSR matrix is %f\n", (nnzb * 1.0) / (mb * nb));

    printf("gpu memory malloc and memcpy...\n");

    HANDLE_ERROR( hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrColInd, nnzb * sizeof(int)) );
    HANDLE_ERROR( hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(bsrRowPtr, hostBsrRowPtr, (size_t)((mb + 1) * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrColInd, hostBsrColInd, (size_t)(nnzb * sizeof(int)), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(bsrVal, hostBsrVal, (size_t)(nnzb * blockDim * blockDim * sizeof(float)), hipMemcpyHostToDevice) );
    
    HANDLE_CUSPARSE_ERROR( hipsparseCreate(&handle), "CUSPARSE Library initialization failed" );
    
    HANDLE_CUSPARSE_ERROR( hipsparseCreateMatDescr(&descr), "BSR Matrix descriptor initialization failed" );
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    printf("prepare y and z...\n");

    yHostPtr = randomDenseMatrix(n, dim);
    zHostPtr = (float*) malloc(m * dim * sizeof(float));

    HANDLE_ERROR( hipMalloc((void**)&y, n * dim * sizeof(float)) );
    HANDLE_ERROR( hipMalloc((void**)&z, m * dim * sizeof(float)) );

    HANDLE_ERROR( hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)), hipMemcpyHostToDevice) );

    // printf("warm up...\n");
    // HANDLE_ERROR( hipMalloc((void**)&y0, n * dim * sizeof(float)) );
    // HANDLE_ERROR( hipMalloc((void**)&z0, m * dim * sizeof(float)) );
    // HANDLE_ERROR( hipMemset((void*)y0, 0, n * dim * sizeof(float)) );
    // HANDLE_ERROR( hipMemset((void*)z0, 0, m * dim * sizeof(float)) );
    // int warnupRounds = 3;
    // for (int i = 0; i < warnupRounds; ++i) {
    //     HANDLE_CUSPARSE_ERROR( hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                           HIPSPARSE_OPERATION_TRANSPOSE, mb, dim, nb, nnzb, &fone, descr, bsrVal,
    //                                           bsrRowPtr, bsrColInd, blockDim, y0, dim, &fzero, z0, m),
    //                            "warmup hipsparseSbsrmm failed" );  
    // }

    printf("hipsparseSbsrmm...\n");
    // float totalTime = 0;
    // int rounds = 10;
    // for (int i = 0; i < rounds; ++i) {
    //     HANDLE_ERROR( hipMemset((void*)z, 0, m * dim * sizeof(float)) );

    //     float time;
    //     hipEvent_t start, stop;
    //     HANDLE_ERROR( hipEventCreate(&start) );
    //     HANDLE_ERROR( hipEventCreate(&stop) );
    //     HANDLE_ERROR( hipEventRecord(start, 0) );

    //     HANDLE_CUSPARSE_ERROR( hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                           HIPSPARSE_OPERATION_TRANSPOSE, mb, dim, nb, nnzb, &fone, descr, bsrVal,
    //                                           bsrRowPtr, bsrColInd, blockDim, y, dim, &fzero, z, m),
    //                            "hipsparseSbsrmm failed" ); 

    //     HANDLE_ERROR( hipEventRecord(stop, 0) );
    //     HANDLE_ERROR( hipEventSynchronize(stop) );
    //     HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    //     printf("[%d] bsrmm cost time:  %3.10f ms \n", i, time);   
    //     totalTime += time;
    // }
    // printf("average bsrmm cost time:  %3.10f ms \n", totalTime / rounds);   


    HANDLE_ERROR( hipMemset((void*)z, 0, m * dim * sizeof(float)) );

    // hipProfilerStart();

    float time;
    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start, 0) );

    HANDLE_CUSPARSE_ERROR( hipsparseSbsrmm(handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_TRANSPOSE, mb, dim, nb, nnzb, &fone, descr, bsrVal,
                                          bsrRowPtr, bsrColInd, blockDim, y, dim, &fzero, z, m),
                           "hipsparseSbsrmm failed" ); 

    HANDLE_ERROR( hipEventRecord(stop, 0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
    printf("bsrmm cost time:  %3.10f ms \n", time);   

    // hipProfilerStop();

    HANDLE_ERROR( hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)), hipMemcpyDeviceToHost) );

    HANDLE_CUSPARSE_ERROR( hipsparseDestroyMatDescr(descr), "Matrix descriptor destruction failed" );
    descr = 0;
    HANDLE_CUSPARSE_ERROR( hipsparseDestroy(handle), "CUSPARSE Library release of resources failed" );
    handle = 0;

    CLEANUP("end");

    return 0;
}