#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <random>
#include <vector>
#include "hip/hip_runtime_api.h"
#include "hipsparse.h"
#include "gespmm_csrmm.h"
#include "load_data.h"
#include "utility.h"

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                     \
  if (!checkError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUDA ERROR");                    \
    exit(-1);                                 \
  }

#define HANDLE_CUSPARSE_ERROR(err)                    \
  if (!checkCusparseError(err, __FILE__, __LINE__)) { \
    CLEANUP("CUSPARSE ERROR");                        \
    exit(-1);                                         \
  }

int main(int argc, char* argv[]) {
  float p = std::stof(argv[1]);
  int dim = std::stoi(argv[2]);
  std::string csrmmImpl(argv[3]);
  printf("p = %f dim = %d csrmmImpl = %s\n", p, dim, csrmmImpl.c_str());

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;

  int m = 2 << 16;
  int n = m;
  int nnz = 0;
  float fzero = 0.0;
  float fone = 1.0;

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;
  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  printf("generate random CSR matrix\n");

  // nnz = randomCSRMatrix(m, n, p, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal,
  // -1, 1, true);
  nnz = readAndFillCSRMatrix(m, n, p, &hostCsrRowPtr, &hostCsrColInd,
                             &hostCsrVal);

  printf("density of CSR matrix is %f\n", ((nnz * 1.0) / m) / n);

  printf("gpu memory malloc and memcpy...\n");

  HANDLE_ERROR(hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&csrColInd, nnz * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&csrVal, nnz * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(csrRowPtr, hostCsrRowPtr,
                          (size_t)((m + 1) * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)),
                          hipMemcpyHostToDevice));

  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&descr));
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  printf("prepare y and z...\n");

  yHostPtr = randomDenseMatrix(n, dim);
  zHostPtr = (float*)malloc(m * dim * sizeof(float));

  HANDLE_ERROR(hipMalloc((void**)&y, n * dim * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&z, m * dim * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)),
                          hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemset((void*)z, 0, m * dim * sizeof(float)));

  printf("hipsparseScsrmm...\n");

  float time;
  hipEvent_t start, stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  hipProfilerStart();

  if (csrmmImpl == "gespmm") {
    gespmm_csrmm<float>(m, dim, csrRowPtr, csrColInd, csrVal, y, z);
  } else if (csrmmImpl == "cusparse") {
    HANDLE_CUSPARSE_ERROR(hipsparseScsrmm2(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        m, dim, n, nnz, &fone, descr, csrVal, csrRowPtr, csrColInd, y, dim,
        &fzero, z, m));
  } else {
    assert(false);
  }

  hipProfilerStop();

  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

  float gflops = (nnz / 1.0e6) * dim / time;
  printf("csrmm cost time: %6.10f ms\nGFLOPs: %6.10f\n", time, gflops);

  HANDLE_ERROR(hipMemcpy(zHostPtr, z, (size_t)(m * dim * sizeof(float)),
                          hipMemcpyDeviceToHost));

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(descr));
  descr = 0;
  HANDLE_CUSPARSE_ERROR(hipsparseDestroy(handle));
  handle = 0;

  CLEANUP("end");

  return 0;
}