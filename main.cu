#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void add(float* x, float * y, float* z, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    float *x, *y, *z;
    x = (float*)malloc(nBytes);
    y = (float*)malloc(nBytes);
    z = (float*)malloc(nBytes);

    for (int i = 0; i < N; ++i) {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    float *d_x, *d_y, *d_z;
    hipMalloc((void**)&d_x, nBytes);
    hipMalloc((void**)&d_y, nBytes);
    hipMalloc((void**)&d_z, nBytes);

    hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
    hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

    add << < gridSize, blockSize >> >(d_x, d_y, d_z, N);

    hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    std::cout << "z0 = " << z[0] << std::endl;
    std::cout << "maximum error: " << maxError << std::endl;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    free(x);
    free(y);
    free(z);

    return 0;
}