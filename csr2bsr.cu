#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <vector>
#include "hipsparse.h"

static void handle_error(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(-1);
  }
}

#define HANDLE_ERROR(err) (handle_error(err, __FILE__, __LINE__))

std::mt19937_64 gen(1234);

template <typename T>
T* vec2ptr(std::vector<T> v) {
  T* ptr = (T*)malloc(v.size() * sizeof(T));
  for (size_t i = 0; i < v.size(); ++i) {
    ptr[i] = v[i];
  }
  return ptr;
}

int randomCSRMatrix(int m, int n, float p, int** hostCsrRowPtr,
                    int** hostCsrColInd, float** hostCsrVal, float minVal = -10,
                    float maxVal = 10) {
  std::uniform_real_distribution<float> flip(0, 1), dist(minVal, maxVal);
  *hostCsrRowPtr = (int*)malloc((m + 1) * sizeof(int));
  int cnt = 0;
  (*hostCsrRowPtr)[0] = cnt;
  std::vector<int> indices;
  std::vector<float> vals;
  for (int i = 1; i <= m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (flip(gen) < p) {
        indices.push_back(j);
        vals.push_back(dist(gen));
        ++cnt;
      }
    }
    (*hostCsrRowPtr)[i] = cnt;
  }
  *hostCsrColInd = vec2ptr(std::move(indices));
  *hostCsrVal = vec2ptr(std::move(vals));

  return cnt;
}

float* randomDenseMatrix(int n, int dim, float minVal = -10,
                         float maxVal = 10) {
  std::uniform_real_distribution<float> dist(minVal, maxVal);
  int sz = n * dim;
  float* ptr = (float*)malloc(sz * sizeof(float));
  for (int i = 0; i < sz; ++i) {
    ptr[i] = dist(gen);
  }
  return ptr;
}

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (z1HostPtr) free(z1HostPtr);         \
    if (z2HostPtr) free(z2HostPtr);         \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (bsrRowPtr) hipFree(bsrRowPtr);     \
    if (bsrColInd) hipFree(bsrColInd);     \
    if (bsrVal) hipFree(bsrVal);           \
    if (y) hipFree(y);                     \
    if (z1) hipFree(z1);                   \
    if (z2) hipFree(z2);                   \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

int main(int argc, char* argv[]) {
  float p = std::stof(argv[1]);
  printf("%f\n", p);

  hipError_t cudaStat1, cudaStat2, cudaStat3;
  hipsparseStatus_t status;
  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t csrDescr = 0, bsrDescr = 0;

  int m = 1000;
  int n = 1200;
  int nnz = 0;
  int blockDim = 2;
  int mb = (m + blockDim - 1) / blockDim;
  int nb = (n + blockDim - 1) / blockDim;
  int nnzb = 0;
  int dim = 100;
  float fzero = 0.0;
  float fone = 1.0;

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;
  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  int* bsrRowPtr = 0;
  int* bsrColInd = 0;
  float* bsrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* z1HostPtr = 0;
  float* z1 = 0;
  float* z2HostPtr = 0;
  float* z2 = 0;

  nnz = randomCSRMatrix(m, n, p, &hostCsrRowPtr, &hostCsrColInd, &hostCsrVal);

  cudaStat1 = hipMalloc((void**)&csrRowPtr, (m + 1) * sizeof(int));
  cudaStat2 = hipMalloc((void**)&csrColInd, nnz * sizeof(int));
  cudaStat3 = hipMalloc((void**)&csrVal, nnz * sizeof(float));
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess ||
      cudaStat3 != hipSuccess) {
    CLEANUP("Device malloc failed (CSR matrix)");
    return 1;
  }

  cudaStat1 =
      hipMemcpy(csrRowPtr, hostCsrRowPtr, (size_t)((m + 1) * sizeof(int)),
                 hipMemcpyHostToDevice);
  cudaStat2 = hipMemcpy(csrColInd, hostCsrColInd, (size_t)(nnz * sizeof(int)),
                         hipMemcpyHostToDevice);
  cudaStat3 = hipMemcpy(csrVal, hostCsrVal, (size_t)(nnz * sizeof(float)),
                         hipMemcpyHostToDevice);
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess ||
      cudaStat3 != hipSuccess) {
    CLEANUP("Memcpy from Host to Device failed (CSR matrix)");
    return 1;
  }

  status = hipsparseCreate(&handle);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("CUSPARSE Library initialization failed");
    return 1;
  }

  status = hipsparseCreateMatDescr(&csrDescr);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("CSR Matrix descriptor initialization failed");
    return 1;
  }
  hipsparseSetMatType(csrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(csrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  status = hipsparseCreateMatDescr(&bsrDescr);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("BSR Matrix descriptor initialization failed");
    return 1;
  }
  hipsparseSetMatType(bsrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(bsrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  cudaStat1 = hipMalloc((void**)&bsrRowPtr, (mb + 1) * sizeof(int));
  if (cudaStat1 != hipSuccess) {
    CLEANUP("Device malloc failed (BSR matrix)");
    return 1;
  }
  status = hipsparseXcsr2bsrNnz(handle, HIPSPARSE_DIRECTION_ROW, m, n, csrDescr,
                               csrRowPtr, csrColInd, blockDim, bsrDescr,
                               bsrRowPtr, &nnzb);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("hipsparseXcsr2bsrNnz failed");
    return 1;
  }
  cudaStat1 = hipMalloc((void**)&bsrColInd, nnzb * sizeof(int));
  cudaStat2 =
      hipMalloc((void**)&bsrVal, nnzb * blockDim * blockDim * sizeof(float));
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
    CLEANUP("Device malloc failed (BSR matrix)");
    return 1;
  }
  status = hipsparseScsr2bsr(handle, HIPSPARSE_DIRECTION_ROW, m, n, csrDescr,
                            csrVal, csrRowPtr, csrColInd, blockDim, bsrDescr,
                            bsrVal, bsrRowPtr, bsrColInd);
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("hipsparseScsr2bsr failed");
    return 1;
  }

  yHostPtr = randomDenseMatrix(n, dim);
  z1HostPtr = (float*)malloc(m * dim * sizeof(float));
  z2HostPtr = (float*)malloc(m * dim * sizeof(float));

  cudaStat1 = hipMalloc((void**)&y, n * dim * sizeof(float));
  cudaStat2 = hipMalloc((void**)&z1, m * dim * sizeof(float));
  cudaStat3 = hipMalloc((void**)&z2, m * dim * sizeof(float));
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess ||
      cudaStat3 != hipSuccess) {
    CLEANUP("Device malloc failed (dense matrix)");
    return 1;
  }

  cudaStat1 = hipMemcpy(y, yHostPtr, (size_t)(n * dim * sizeof(float)),
                         hipMemcpyHostToDevice);
  if (cudaStat1 != hipSuccess) {
    CLEANUP("Memcpy from Host to Device failed (y)");
    return 1;
  }

  cudaStat1 = hipMemset((void*)z1, 0, m * dim * sizeof(float));
  cudaStat2 = hipMemset((void*)z2, 0, m * dim * sizeof(float));
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
    CLEANUP("Memset on Device failed (z)");
    return 1;
  }

  float time1, time2;
  hipEvent_t start1, stop1, start2, stop2;

  HANDLE_ERROR(hipEventCreate(&start1));
  HANDLE_ERROR(hipEventCreate(&stop1));
  HANDLE_ERROR(hipEventRecord(start1, 0));

  status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, dim, n,
                          nnz, &fone, csrDescr, csrVal, csrRowPtr, csrColInd, y,
                          n, &fzero, z1, m);

  HANDLE_ERROR(hipEventRecord(stop1, 0));
  HANDLE_ERROR(hipEventSynchronize(stop1));
  HANDLE_ERROR(hipEventElapsedTime(&time1, start1, stop1));

  printf("csrmm cost time:  %3.10f ms \n", time1);

  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("csrmm failed");
    return 1;
  }

  HANDLE_ERROR(hipEventCreate(&start2));
  HANDLE_ERROR(hipEventCreate(&stop2));
  HANDLE_ERROR(hipEventRecord(start2, 0));

  status = hipsparseSbsrmm(
      handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      HIPSPARSE_OPERATION_NON_TRANSPOSE, mb, dim, nb, nnzb, &fone, bsrDescr,
      bsrVal, bsrRowPtr, bsrColInd, blockDim, y, n, &fzero, z2, m);

  HANDLE_ERROR(hipEventRecord(stop2, 0));
  HANDLE_ERROR(hipEventSynchronize(stop2));
  HANDLE_ERROR(hipEventElapsedTime(&time2, start2, stop2));

  printf("bsrmm cost time:  %3.10f ms \n", time2);

  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("bsrmm failed");
    return 1;
  }

  cudaStat1 = hipMemcpy(z1HostPtr, z1, (size_t)(m * dim * sizeof(float)),
                         hipMemcpyDeviceToHost);
  cudaStat2 = hipMemcpy(z2HostPtr, z2, (size_t)(m * dim * sizeof(float)),
                         hipMemcpyDeviceToHost);
  if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess) {
    CLEANUP("Memcpy from Device to Host failed (z)");
    return 1;
  }

  status = hipsparseDestroyMatDescr(csrDescr);
  csrDescr = 0;
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("CSR matrix descriptor destruction failed");
    return 1;
  }

  status = hipsparseDestroyMatDescr(bsrDescr);
  bsrDescr = 0;
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("BSR matrix descriptor destruction failed");
    return 1;
  }

  status = hipsparseDestroy(handle);
  handle = 0;
  if (status != HIPSPARSE_STATUS_SUCCESS) {
    CLEANUP("CUSPARSE Library release of resources failed");
    return 1;
  }

  bool flag = true;
  for (int i = 0; i < m * dim; ++i) {
    float error = fabs(z1HostPtr[i] - z2HostPtr[i]);
    if (error > 0.01) {
      printf("inconsistent result: %d %f", i, error);
      flag = false;
      break;
    }
  }

  if (flag) {
    printf("\nsame result\n");
  } else {
    printf("\ninconsistent result\n");
  }
  CLEANUP("end");
}