#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <array>
#include <fstream>
#include <iostream>
#include <random>
#include <string>
#include <vector>
#include "hipsparse.h"
#include "load_data.h"
#include "rocsparse_bsrmm.h"
#include "utility.h"

std::mt19937_64 gen(1234);

std::vector<std::vector<int>> csr2adj(const std::string& indptr_file,
                                      const std::string& indices_file, int& n,
                                      int& nnz) {
  std::fstream s1(indptr_file, std::ios::in);
  std::fstream s2(indices_file, std::ios::in);

  std::vector<int> indptr, indices;

  int xx;
  s1 >> xx;
  n = xx - 1;
  for (int i = 0; i <= n; ++i) {
    s1 >> xx;
    indptr.push_back(xx);
  }
  nnz = indptr[n] - indptr[0];
  s2 >> xx;
  assert(xx == nnz);
  for (int i = 0; i < nnz; ++i) {
    s2 >> xx;
    indices.push_back(xx);
  }

  assert(indptr[0] == 0);
  std::vector<std::vector<int>> edges(n);
  for (int i = 0; i < n; ++i) {
    int start = indptr[i], end = indptr[i + 1];
    for (int j = start; j < end; ++j) {
      edges[i].push_back(indices[j]);
    }
  }

  return edges;
}

void divide_matrix(const std::vector<std::vector<int>>& edges,
                   std::vector<int>& csr_row_ptr, std::vector<int>& csr_col_ind,
                   std::vector<int>& bsr_row_ptr, std::vector<int>& bsr_col_ind,
                   std::vector<float>& bsr_val, int n, int bsize,
                   float density) {
  int nb = (n + bsize - 1) / bsize;
  int bnum = bsize * bsize;
  std::vector<int> counts(nb, 0);
  std::vector<int> flags(nb, -1);

  csr_row_ptr.push_back(0);
  bsr_row_ptr.push_back(0);

  for (int x1 = 0; x1 < nb; ++x1) {
    std::fill(counts.begin(), counts.end(), 0);
    std::fill(flags.begin(), flags.end(), -1);

    for (int x2 = 0; x2 < bsize; ++x2) {
      int x = x1 * bsize + x2;
      if (x >= n) {
        break;
      }
      const std::vector<int>& ys = edges[x];
      // for (int y : ys) {
      for (int i = 0; i < ys.size(); ++i) {
        // if (i >= 1 && ys[i] == ys[i - 1]) {
        //     std::cout << "????" << std::endl;
        // }
        int y = ys[i];
        ++counts[y / bsize];
      }
    }

    int bsr_cnt = 0;
    for (int i = 0; i < nb; ++i) {
      float occupy = (counts[i] * 1.0) / bnum;
      // if (counts[i] > bnum) {
      //     std::cout << counts[i] << " oooops" << std::endl;
      // }
      if (occupy >= density) {
        bsr_col_ind.push_back(i);
        flags[i] = bsr_cnt;
        ++bsr_cnt;
      }
    }

    int bsr_row_val = bsr_row_ptr.back() + bsr_cnt;
    bsr_row_ptr.push_back(bsr_row_val);

    std::vector<float> vals(bsr_cnt * bnum, 0);
    for (int x2 = 0; x2 < bsize; ++x2) {
      int x = x1 * bsize + x2;
      if (x >= n) {
        break;
      }
      int csr_cnt = 0;
      const std::vector<int>& ys = edges[x];
      for (int y : ys) {
        int y1 = y / bsize, y2 = y % bsize;
        int ith = flags[y1];
        if (ith == -1) {
          csr_col_ind.push_back(y);
          ++csr_cnt;
        } else {
          vals[ith * bnum + x2 * bsize + y2] = 1;
        }
      }
      int csr_row_val = csr_row_ptr.back() + csr_cnt;
      csr_row_ptr.push_back(csr_row_val);
    }

    for (float x : vals) {
      bsr_val.push_back(x);
    }
  }
}

// template <class T>
// void dump_vec(const std::vector<T>& vec, const std::string& name) {
//   std::cout << name << ": ";
//   for (T x : vec) {
//     std::cout << x << " ";
//   }
//   std::cout << std::endl;
// }

// void test_divide_matrix() {
//   int n = 4, bsize = 2;
//   float density = 0.6;
//   std::vector<std::vector<int>> edges = {{
//                                              1,
//                                          },
//                                          {
//                                              2,
//                                          },
//                                          {1, 3},
//                                          {0, 1, 2}};
//   std::vector<int> csr_row_ptr, csr_col_ind, bsr_row_ptr, bsr_col_ind;
//   std::vector<float> bsr_val;

//   divide_matrix(edges, csr_row_ptr, csr_col_ind, bsr_row_ptr, bsr_col_ind,
//                 bsr_val, n, bsize, density);

//   dump_vec(csr_row_ptr, "csr_row_ptr");
//   dump_vec(csr_col_ind, "csr_col_ind");

//   dump_vec(bsr_row_ptr, "bsr_row_ptr");
//   dump_vec(bsr_col_ind, "bsr_col_ind");
//   dump_vec(bsr_val, "bsr_val");
// }

#define CLEANUP(s)                          \
  do {                                      \
    printf("%s\n", s);                      \
    if (hostCsrRowPtr) free(hostCsrRowPtr); \
    if (hostCsrColInd) free(hostCsrColInd); \
    if (hostCsrVal) free(hostCsrVal);       \
    if (hostBsrRowPtr) free(hostBsrRowPtr); \
    if (hostBsrColInd) free(hostBsrColInd); \
    if (hostBsrVal) free(hostBsrVal);       \
    if (yHostPtr) free(yHostPtr);           \
    if (zHostPtr) free(zHostPtr);           \
    if (csrRowPtr) hipFree(csrRowPtr);     \
    if (csrColInd) hipFree(csrColInd);     \
    if (csrVal) hipFree(csrVal);           \
    if (bsrRowPtr) hipFree(bsrRowPtr);     \
    if (bsrColInd) hipFree(bsrColInd);     \
    if (bsrVal) hipFree(bsrVal);           \
    if (y) hipFree(y);                     \
    if (z) hipFree(z);                     \
    hipDeviceReset();                      \
    fflush(stdout);                         \
  } while (0)

#define HANDLE_ERROR(err)                                            \
  if (err != hipSuccess) {                                          \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP("");                                                     \
    exit(-1);                                                        \
  }

#define HANDLE_CUSPARSE_ERROR(err)                                   \
  if (err != HIPSPARSE_STATUS_SUCCESS) {                              \
    printf("error occurred in %s at line %d\n", __FILE__, __LINE__); \
    CLEANUP("");                                                     \
    exit(-1);                                                        \
  }

int main(int argc, char* argv[]) {
  std::string prefix = "tmp/" + std::string(argv[1]);
  std::cout << prefix << std::endl;
  int bsize = std::stoi(argv[2]);
  int dim = std::stoi(argv[3]);
  std::string bsrmmImpl(argv[4]);
  int transposeB = std::stoi(argv[5]);
  float density = std::stof(argv[6]);
  
  int n;
  int nnz;
  std::string indptr_file = prefix + "_indptr.txt";
  std::string indices_file = prefix + "_indices.txt";
  std::cout << "csr to adj..." << std::endl;
  std::vector<std::vector<int>> edges = csr2adj(indptr_file, indices_file, n, nnz);

  int bnum = bsize * bsize;
  int nb = (n + bsize - 1) / bsize;
  int n1 = nb * bsize;
  assert(n1 >= n);
  float alpha = 1.0;
  float beta = 1.0;

  hipsparseOperation_t transB;
  int ldb;
  if (transposeB == 0) {
    transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    ldb = n;
  } else if (transposeB == 1) {
    transB = HIPSPARSE_OPERATION_TRANSPOSE;
    ldb = dim;
  } else {
    assert(false);
  }

  std::vector<int> csr_row_ptr, csr_col_ind, bsr_row_ptr, bsr_col_ind;
  std::vector<float> bsr_val;

  std::cout << "divide matrix..." << std::endl;

  divide_matrix(edges, csr_row_ptr, csr_col_ind, bsr_row_ptr, bsr_col_ind,
                bsr_val, n, bsize, density);

  assert(csr_row_ptr.size() == n + 1);
  assert(csr_row_ptr[0] == 0);
  assert(csr_row_ptr[n] == csr_col_ind.size());
  assert(bsr_row_ptr.size() == nb + 1);
  assert(bsr_row_ptr[0] == 0);
  assert(bsr_row_ptr[nb] == bsr_col_ind.size());
  assert(bsr_col_ind.size() * bnum == bsr_val.size());

  int csrNnz = csr_col_ind.size();
  int bsrNnzb = bsr_col_ind.size();

  printf("csr nnz = %d    bsr nnzb = %d\n", csrNnz, bsrNnzb);

  int* hostCsrRowPtr = 0;
  int* hostCsrColInd = 0;
  float* hostCsrVal = 0;
  int* csrRowPtr = 0;
  int* csrColInd = 0;
  float* csrVal = 0;

  int* hostBsrRowPtr = 0;
  int* hostBsrColInd = 0;
  float* hostBsrVal = 0;
  int* bsrRowPtr = 0;
  int* bsrColInd = 0;
  float* bsrVal = 0;

  float* yHostPtr = 0;
  float* y = 0;
  float* zHostPtr = 0;
  float* z = 0;

  std::cout << "vec2ptr..." << std::endl;

  hostCsrRowPtr = vec2ptr<int>(csr_row_ptr);
  hostCsrColInd = vec2ptr<int>(csr_col_ind);
  hostCsrVal = (float*)malloc(csrNnz * sizeof(float));
  for (int i = 0; i < csrNnz; ++i) {
    hostCsrVal[i] = 1.0;
  }

  hostBsrRowPtr = vec2ptr<int>(bsr_row_ptr);
  hostBsrColInd = vec2ptr<int>(bsr_col_ind);
  hostBsrVal = vec2ptr<float>(bsr_val);

  std::cout << "gpu memory malloc and memcpy..." << std::endl;

  HANDLE_ERROR(hipMalloc((void**)&csrRowPtr, (n + 1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&csrColInd, csrNnz * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&csrVal, csrNnz * sizeof(float)));

  HANDLE_ERROR(hipMalloc((void**)&bsrRowPtr, (nb + 1) * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&bsrColInd, bsrNnzb * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&bsrVal, bsrNnzb * bnum * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(csrRowPtr, hostCsrRowPtr,
                          (size_t)((n + 1) * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(csrColInd, hostCsrColInd,
                          (size_t)(csrNnz * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(csrVal, hostCsrVal, (size_t)(csrNnz * sizeof(float)),
                          hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemcpy(bsrRowPtr, hostBsrRowPtr,
                          (size_t)((nb + 1) * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(bsrColInd, hostBsrColInd,
                          (size_t)(bsrNnzb * sizeof(int)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(bsrVal, hostBsrVal,
                          (size_t)(bsrNnzb * bnum * sizeof(float)),
                          hipMemcpyHostToDevice));

  std::cout << "prepare y and z..." << std::endl;

  yHostPtr = randomDenseMatrix(n1, dim);
  zHostPtr = (float*)malloc(n1 * dim * sizeof(float));

  HANDLE_ERROR(hipMalloc((void**)&y, n1 * dim * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&z, n1 * dim * sizeof(float)));

  HANDLE_ERROR(hipMemcpy(y, yHostPtr, (size_t)(n1 * dim * sizeof(float)),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset((void*)z, 0, n1 * dim * sizeof(float)));

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t csrDescr = 0, bsrDescr = 0;

  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&csrDescr));
  hipsparseSetMatType(csrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(csrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&bsrDescr));
  hipsparseSetMatType(bsrDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(bsrDescr, HIPSPARSE_INDEX_BASE_ZERO);

  float time1, time2;
  hipEvent_t start1, stop1, start2, stop2;
  HANDLE_ERROR(hipEventCreate(&start1));
  HANDLE_ERROR(hipEventCreate(&stop1));
  HANDLE_ERROR(hipEventRecord(start1, 0));

  HANDLE_CUSPARSE_ERROR(hipsparseScsrmm2(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, transB, n,
      dim, n, csrNnz, &alpha, csrDescr, csrVal, csrRowPtr, csrColInd, y, ldb,
      &beta, z, n1));

  HANDLE_ERROR(hipEventRecord(stop1, 0));
  HANDLE_ERROR(hipEventSynchronize(stop1));
  HANDLE_ERROR(hipEventElapsedTime(&time1, start1, stop1));

  HANDLE_ERROR(hipEventCreate(&start2));
  HANDLE_ERROR(hipEventCreate(&stop2));
  HANDLE_ERROR(hipEventRecord(start2, 0));

  if (bsrmmImpl == "rocsparse") {
    HANDLE_CUSPARSE_ERROR(rocsparse_bsrmm_template<float>(
        handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        transB, nb, dim, nb, bsrNnzb, alpha, bsrDescr,
        bsrVal, bsrRowPtr, bsrColInd, bsize, y, ldb, beta, z, n1));
  } else if (bsrmmImpl == "cusparse") { 
    HANDLE_CUSPARSE_ERROR(hipsparseSbsrmm(
        handle, HIPSPARSE_DIRECTION_ROW, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        transB, nb, dim, nb, bsrNnzb, &alpha, bsrDescr,
        bsrVal, bsrRowPtr, bsrColInd, bsize, y, ldb, &beta, z, n1));
  } else {
    assert(false);
  }

  HANDLE_ERROR(hipEventRecord(stop2, 0));
  HANDLE_ERROR(hipEventSynchronize(stop2));
  HANDLE_ERROR(hipEventElapsedTime(&time2, start2, stop2));

  printf("csrmm cost time:  %3.10f ms \n", time1);
  printf("bsrmm cost time:  %3.10f ms \n", time2);
  printf("total cost time:  %3.10f ms \n", time1 + time2);
  printf("%3.5f+%3.5f=%3.5f\n", time1, time2, time1 + time2);

  HANDLE_ERROR(hipMemcpy(zHostPtr, z, (size_t)(n1 * dim * sizeof(float)),
                          hipMemcpyDeviceToHost));

  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(bsrDescr));
  bsrDescr = 0;
  HANDLE_CUSPARSE_ERROR(hipsparseDestroyMatDescr(csrDescr));
  csrDescr = 0;
  HANDLE_CUSPARSE_ERROR(hipsparseDestroy(handle));
  handle = 0;

  CLEANUP("end");

  return 0;
}
